#include <hip/hip_runtime.h>

/*Lx2Cuda performs the 2-D convolution of matrices A and row vector B*/
__global__ void Lx2(const float *d_in,float *d_out,int numRows,int numCols, float *mask)
{

    //Calculate the row # of the d_in and d_out element to process
    int Col = blockIdx.y*blockDim.y + threadIdx.y;
    
    //Calculate the column # of the d_in and d_out element to process
    int Row = blockIdx.x*blockDim.x + threadIdx.x;
    
    // each thread computes one elements 
    if ((4<Row) && (Row<numRows-5) && (4<Col) && (Col<numCols-5)){
        d_out[Col*numRows+Row]=mask[0]*d_in[Col*numRows+Row]+
                mask[1]*(d_in[(Col-1)*numRows+Row]+d_in[(Col+1)*numRows+Row])+
                mask[2]*(d_in[(Col-2)*numRows+Row]+d_in[(Col+2)*numRows+Row])+
                mask[3]*(d_in[(Col-3)*numRows+Row]+d_in[(Col+3)*numRows+Row])+
                mask[4]*(d_in[(Col-4)*numRows+Row]+d_in[(Col+4)*numRows+Row])+
                mask[5]*(d_in[(Col-5)*numRows+Row]+d_in[(Col+5)*numRows+Row]);
    }
}

/*Lz2 performs the 2-D convolution of matrices A and column vector B*/
__global__ void Lz2(const float *d_in,float *d_out,int numRows,int numCols, float *mask)
{

    //Calculate the row # of the d_in and d_out element to process
    int Col = blockIdx.y*blockDim.y + threadIdx.y;
    
    //Calculate the column # of the d_in and d_out element to process
    int Row = blockIdx.x*blockDim.x + threadIdx.x;
    
   
    // each thread computes one elements 
    if ((4<Row) && (Row<numRows-5) && (4<Col) && (Col<numCols-5)){
        int Loc = Col*numRows+Row;
        d_out[Loc]=mask[0]*d_in[Loc]+
                mask[1]*(d_in[Loc-1]+d_in[Loc+1])+
                mask[2]*(d_in[Loc-2]+d_in[Loc+2])+
                mask[3]*(d_in[Loc-3]+d_in[Loc+3])+
                mask[4]*(d_in[Loc-4]+d_in[Loc+4])+
                mask[5]*(d_in[Loc-5]+d_in[Loc+5]);
    }
}

/*Lz1 performs the 2-D convolution of matrices A and column vector C1*/
__global__ void Lz1(const float *d_in,float *d_out, int numRows,int numCols, float *mask)
{

    //Calculate the row # of the d_in and d_out element to process
    int Col = blockIdx.y*blockDim.y + threadIdx.y;
    
    //Calculate the column # of the d_in and d_out element to process
    int Row = blockIdx.x*blockDim.x + threadIdx.x;
    
    // each thread computes one elements 
    if ((5<Row) && (Row<numRows-6) && (5<Col) && (Col<numCols-6)){
        int Loc = Col*numRows+Row;
        d_out[Loc]=
                mask[0]*(d_in[Loc+1]-d_in[Loc-1])+
                mask[1]*(d_in[Loc+2]-d_in[Loc-2])+
                mask[2]*(d_in[Loc+3]-d_in[Loc-3])+
                mask[3]*(d_in[Loc+4]-d_in[Loc-4])+
                mask[4]*(d_in[Loc+5]-d_in[Loc-5])+
                mask[5]*(d_in[Loc+6]-d_in[Loc-6]);
    }
}

/*Lx1 performs the 2-D convolution of matrices A and row vector C1*/
__global__ void Lx1(const float *d_in,float *d_out, int numRows,int numCols, float *mask)
{

    //Calculate the row # of the d_in and d_out element to process
    int Col = blockIdx.y*blockDim.y + threadIdx.y;
    
    //Calculate the column # of the d_in and d_out element to process
    int Row = blockIdx.x*blockDim.x + threadIdx.x;
    
    // each thread computes one elements 
    if ((5<Row) && (Row<numRows-6) && (5<Col) && (Col<numCols-6)){
        d_out[Col*numRows+Row]=
                mask[0]*(d_in[(Col+1)*numRows+Row]-d_in[(Col-1)*numRows+Row])+
                mask[1]*(d_in[(Col+2)*numRows+Row]-d_in[(Col-2)*numRows+Row])+
                mask[2]*(d_in[(Col+3)*numRows+Row]-d_in[(Col-3)*numRows+Row])+
                mask[3]*(d_in[(Col+4)*numRows+Row]-d_in[(Col-4)*numRows+Row])+
                mask[4]*(d_in[(Col+5)*numRows+Row]-d_in[(Col-5)*numRows+Row])+
                mask[5]*(d_in[(Col+6)*numRows+Row]-d_in[(Col-6)*numRows+Row]);
    }

}

/*sbLx performs the 2-D convolution of matrices A and row vector S1*/
__global__ void sbLx(const float *d_in,float *d_out, int numRows,int numCols, float *mask)
{

    //Calculate the row # of the d_in and d_out element to process
    int Col = blockIdx.y*blockDim.y + threadIdx.y;
    
    //Calculate the column # of the d_in and d_out element to process
    int Row = blockIdx.x*blockDim.x + threadIdx.x;
    
    // each thread computes one elements 
    if ((5<Row) && (Row<numRows-6) && (5<Col) && (Col<numCols-6)){
        d_out[Col*numRows+Row]=
                mask[0]*(d_in[(Col+1)*numRows+Row]-d_in[(Col-0)*numRows+Row])+
                mask[1]*(d_in[(Col+2)*numRows+Row]-d_in[(Col-1)*numRows+Row])+
                mask[2]*(d_in[(Col+3)*numRows+Row]-d_in[(Col-2)*numRows+Row])+
                mask[3]*(d_in[(Col+4)*numRows+Row]-d_in[(Col-3)*numRows+Row])+
                mask[4]*(d_in[(Col+5)*numRows+Row]-d_in[(Col-4)*numRows+Row])+
                mask[5]*(d_in[(Col+6)*numRows+Row]+d_in[(Col-5)*numRows+Row]);
    }

}

/*sfLx performs the 2-D convolution of matrices A and row vector S1*/
__global__ void sfLx(const float *d_in,float *d_out, int numRows,int numCols, float *mask)
{

    //Calculate the row # of the d_in and d_out element to process
    int Col = blockIdx.y*blockDim.y + threadIdx.y;
    
    //Calculate the column # of the d_in and d_out element to process
    int Row = blockIdx.x*blockDim.x + threadIdx.x;
    
    // each thread computes one elements 
    if ((5<Row) && (Row<numRows-6) && (5<Col) && (Col<numCols-6)){
        d_out[Col*numRows+Row]=
                mask[0]*(d_in[(Col+0)*numRows+Row]-d_in[(Col-1)*numRows+Row])+
                mask[1]*(d_in[(Col+1)*numRows+Row]-d_in[(Col-2)*numRows+Row])+
                mask[2]*(d_in[(Col+2)*numRows+Row]-d_in[(Col-3)*numRows+Row])+
                mask[3]*(d_in[(Col+3)*numRows+Row]-d_in[(Col-4)*numRows+Row])+
                mask[4]*(d_in[(Col+4)*numRows+Row]-d_in[(Col-5)*numRows+Row])+
                mask[5]*(d_in[(Col+5)*numRows+Row]+d_in[(Col-6)*numRows+Row]);
    }
}
/*sbLz performs the 2-D convolution of matrices A and column vector S1*/
__global__ void sbLz(const float *d_in,float *d_out, int numRows,int numCols, float *mask)
{

    //Calculate the row # of the d_in and d_out element to process
    int Col = blockIdx.y*blockDim.y + threadIdx.y;
    
    //Calculate the column # of the d_in and d_out element to process
    int Row = blockIdx.x*blockDim.x + threadIdx.x;

    // each thread computes one elements 
    if ((5<Row) && (Row<numRows-6) && (5<Col) && (Col<numCols-6)){
        int Loc = Col*numRows+Row;
        d_out[Col*numRows+Row]=
                mask[0]*(d_in[Loc+1]-d_in[Loc-0])+
                mask[1]*(d_in[Loc+2]-d_in[Loc-1])+
                mask[2]*(d_in[Loc+3]-d_in[Loc-2])+
                mask[3]*(d_in[Loc+4]-d_in[Loc-3])+
                mask[4]*(d_in[Loc+5]-d_in[Loc-4])+
                mask[5]*(d_in[Loc+6]+d_in[Loc-5]);
    }

}
/*sfLz performs the 2-D convolution of matrices A and column vector S1*/
__global__ void sfLz(const float *d_in,float *d_out, int numRows,int numCols, float *mask)
{

    //Calculate the row # of the d_in and d_out element to process
    int Col = blockIdx.y*blockDim.y + threadIdx.y;
    
    //Calculate the column # of the d_in and d_out element to process
    int Row = blockIdx.x*blockDim.x + threadIdx.x;
    
    // each thread computes one elements 
    if ((5<Row) && (Row<numRows-6) && (5<Col) && (Col<numCols-6)){
        int Loc = Col*numRows+Row;
        d_out[Loc]=
                mask[0]*(d_in[Loc+0]-d_in[Loc-1])+
                mask[1]*(d_in[Loc+1]-d_in[Loc-2])+
                mask[2]*(d_in[Loc+2]-d_in[Loc-3])+
                mask[3]*(d_in[Loc+3]-d_in[Loc-4])+
                mask[4]*(d_in[Loc+4]-d_in[Loc-5])+
                mask[5]*(d_in[Loc+5]+d_in[Loc-6]);
    }

}
/*rsgffd performs the 2-D forward rotated staggered-grid finite difference*/
__global__ void rsgffd(const float *d_in,float *d_outx,float *d_outz,const int numRows,int numCols,float *mask)
{
    //Calculate the row # of the d_in and d_out element to process
    int Col = blockIdx.y*blockDim.y + threadIdx.y;
    
    //Calculate the column # of the d_in and d_out element to process
    int Row = blockIdx.x*blockDim.x + threadIdx.x;
    
    // each thread computes one elements 
    if ((5<Row) && (Row<numRows-6) && (5<Col) && (Col<numCols-6)){

        d_outx[Col*numRows+Row]=
                    mask[0]*(d_in[(Col+1)*numRows+Row-0]-d_in[(Col-0)*numRows+Row+1])+
                    mask[1]*(d_in[(Col+2)*numRows+Row-1]-d_in[(Col-1)*numRows+Row+2])+
                    mask[2]*(d_in[(Col+3)*numRows+Row-2]-d_in[(Col-2)*numRows+Row+3])+
                    mask[3]*(d_in[(Col+4)*numRows+Row-3]-d_in[(Col-3)*numRows+Row+4])+
                    mask[4]*(d_in[(Col+5)*numRows+Row-4]-d_in[(Col-4)*numRows+Row+5])+
                    mask[5]*(d_in[(Col+6)*numRows+Row-5]-d_in[(Col-5)*numRows+Row+6]);
        d_outz[Col*numRows+Row]=
                    mask[0]*(d_in[(Col+1)*numRows+Row+1]-d_in[(Col-0)*numRows+Row-0])+
                    mask[1]*(d_in[(Col+2)*numRows+Row+2]-d_in[(Col-1)*numRows+Row-1])+
                    mask[2]*(d_in[(Col+3)*numRows+Row+3]-d_in[(Col-2)*numRows+Row-2])+
                    mask[3]*(d_in[(Col+4)*numRows+Row+4]-d_in[(Col-3)*numRows+Row-3])+
                    mask[4]*(d_in[(Col+5)*numRows+Row+5]-d_in[(Col-4)*numRows+Row-4])+
                    mask[5]*(d_in[(Col+6)*numRows+Row+6]-d_in[(Col-5)*numRows+Row-5]);
    }
}

/*rsgbfd performs the 2-D backward rotated staggered-grid finite difference*/
__global__ void rsgbfd(const float *d_in,float *d_outx,float *d_outz,int numRows,int numCols,float *mask)
{
    //Calculate the row # of the d_in and d_out element to process
    int Col = blockIdx.y*blockDim.y + threadIdx.y;
    
    //Calculate the column # of the d_in and d_out element to process
    int Row = blockIdx.x*blockDim.x + threadIdx.x;
    
    // each thread computes one elements 
    if ((5<Row) && (Row<numRows-6) && (5<Col) && (Col<numCols-6)){
        d_outx[Col*numRows+Row]= 
                    mask[0]*(d_in[(Col+0)*numRows+Row-1]-d_in[(Col-1)*numRows+Row+0])+
                    mask[1]*(d_in[(Col+1)*numRows+Row-2]-d_in[(Col-2)*numRows+Row+1])+
                    mask[2]*(d_in[(Col+2)*numRows+Row-3]-d_in[(Col-3)*numRows+Row+2])+
                    mask[3]*(d_in[(Col+3)*numRows+Row-4]-d_in[(Col-4)*numRows+Row+3])+
                    mask[4]*(d_in[(Col+4)*numRows+Row-5]-d_in[(Col-5)*numRows+Row+4])+
                    mask[5]*(d_in[(Col+5)*numRows+Row-6]-d_in[(Col-6)*numRows+Row+5]);
        d_outz[Col*numRows+Row]= 
                    mask[0]*(d_in[(Col+0)*numRows+Row+0]-d_in[(Col-1)*numRows+Row-1])+
                    mask[1]*(d_in[(Col+1)*numRows+Row+1]-d_in[(Col-2)*numRows+Row-2])+
                    mask[2]*(d_in[(Col+2)*numRows+Row+2]-d_in[(Col-3)*numRows+Row-3])+
                    mask[3]*(d_in[(Col+3)*numRows+Row+3]-d_in[(Col-4)*numRows+Row-4])+
                    mask[4]*(d_in[(Col+4)*numRows+Row+4]-d_in[(Col-5)*numRows+Row-5])+
                    mask[5]*(d_in[(Col+5)*numRows+Row+5]-d_in[(Col-6)*numRows+Row-6]);
    }
}




